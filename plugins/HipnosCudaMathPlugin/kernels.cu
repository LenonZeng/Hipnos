#include "hip/hip_runtime.h"
#include <hip/hip_complex.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/functional.h>

#include <iostream>

struct complex_exp_functor : public thrust::unary_function<hipDoubleComplex,hipDoubleComplex>
{
    __host__ __device__ hipDoubleComplex operator()(const hipDoubleComplex &arg) const
    {
        //exp(z) = exp(x) * (cos(y) + i * sin(y))
        double e = exp(arg.x);
        double s, c;
        sincos(arg.y, &s, &c);
        return make_hipDoubleComplex(c * e, s * e);
    }
};

struct comlex_mult_functor : public thrust::binary_function<hipDoubleComplex,hipDoubleComplex,hipDoubleComplex>
{
    __host__ __device__ hipDoubleComplex operator()(const hipDoubleComplex &lhs, const hipDoubleComplex &rhs) const
    {
        return make_hipDoubleComplex((lhs.x * rhs.x) - (lhs.y * rhs.y), (lhs.x * rhs.y) + (lhs.y * rhs.x));
    }
};

struct complex_sequence_functor
{
    const double init;
    const double step;

    complex_sequence_functor(double _init, double _step)
        : init(_init), step(_step) {}

    __host__ __device__ hipDoubleComplex operator()(const int i) const
    {
        return make_hipDoubleComplex(init + step * i, 0);
    }
};

extern "C"
hipError_t cuda_exp(hipDoubleComplex* data, int size)
{
    thrust::device_ptr<hipDoubleComplex> dev_ptr(data);
    thrust::transform(dev_ptr, dev_ptr + size, dev_ptr, complex_exp_functor());
    return hipGetLastError();
}

extern "C"
hipError_t cuda_mult_inplace(hipDoubleComplex* data1, hipDoubleComplex* data2, int size)
{
    thrust::device_ptr<hipDoubleComplex> dev_ptr1(data1);
    thrust::device_ptr<hipDoubleComplex> dev_ptr2(data2);
    thrust::transform(dev_ptr1, dev_ptr1 + size, dev_ptr2, dev_ptr1, comlex_mult_functor());
    return hipGetLastError();
}

extern "C"
hipError_t cuda_mult(hipDoubleComplex* data1, hipDoubleComplex* data2, hipDoubleComplex* result, int size)
{
    thrust::device_ptr<hipDoubleComplex> dev_ptr1(data1);
    thrust::device_ptr<hipDoubleComplex> dev_ptr2(data2);
    thrust::device_ptr<hipDoubleComplex> dev_ptr3(result);
    thrust::transform(dev_ptr1, dev_ptr1 + size, dev_ptr2, dev_ptr3, comlex_mult_functor());
    return hipGetLastError();
}

extern "C"
hipError_t cuda_pow(hipDoubleComplex* data1, int size, int pow)
{
    thrust::device_ptr<hipDoubleComplex> dev_ptr(data1);
    while(pow > 1)
    {
        thrust::transform(dev_ptr, dev_ptr + size, dev_ptr, dev_ptr, comlex_mult_functor());
        pow--;
    }
    return hipGetLastError();
}

extern "C"
hipError_t cuda_for_matrix(hipDoubleComplex* fx, hipDoubleComplex* fy, int rows, int cols, double stepx, double stepy)
{
    thrust::device_ptr<hipDoubleComplex> dev_ptr_fx(fx);
    thrust::device_ptr<hipDoubleComplex> dev_ptr_fy(fy);
    for(int c = 0; c < cols; c++)
    {
        thrust::fill(dev_ptr_fx + c*rows, dev_ptr_fx + c*rows + rows, make_hipDoubleComplex(stepx * (double)(c-cols/2), 0.0));
        thrust::counting_iterator<int> iter = thrust::make_counting_iterator(0);
        thrust::transform(iter, iter + rows, dev_ptr_fy + c*rows, complex_sequence_functor(stepy * (double)(-rows/2), stepy));
    }
    return hipGetLastError();
}
